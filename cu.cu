#include <hip/hip_runtime.h>
#include <iostream>

#define N 10  // Размер таблицы

__global__ void initializeTable(int *table, int size) {
    int i = threadIdx.x;
    if (i < size) {
        table[i] = i * i;  // Пример инициализации: квадрат индекса
    }
}

int main() {
    int h_table[N];         // Таблица на хосте (CPU)
    int *d_table;           // Таблица на устройстве (GPU)

    // Выделение памяти на устройстве
    hipMalloc(&d_table, N * sizeof(int));

    // Инициализация таблицы на GPU
    initializeTable<<<1, N>>>(d_table, N);

    // Копирование данных обратно на хост
    hipMemcpy(h_table, d_table, N * sizeof(int), hipMemcpyDeviceToHost);

    // Вывод результата
    std::cout << "Таблица (квадраты чисел):" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_table[i] << " ";
    }

    // Освобождение памяти
    hipFree(d_table);

    return 0;
}

